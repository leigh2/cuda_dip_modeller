
#include <hip/hip_runtime.h>
#include <float.h>

// fill the stellar grid array
__global__ void make_mugrid(
    const int n_elem_per_dim,
    const int overbin,
    double * xgrid,
    double * ygrid,
    double * mugrid
){
    // thread location, return if out of bounds
    const int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (x_idx >= n_elem_per_dim || y_idx >= n_elem_per_dim) return;

    // 2d array pointer
    const int ptr2d = x_idx + n_elem_per_dim*y_idx;

    // populate xgrid and ygrid arrays
    double xloc = xgrid[ptr2d] = 2.0 * (x_idx + 0.5) / n_elem_per_dim - 1.0;
    double yloc = ygrid[ptr2d] = 2.0 * (y_idx + 0.5) / n_elem_per_dim - 1.0;

    // distance from center of the star squared
    double d2 = xloc * xloc + yloc * yloc;

    // populate the mu array
    mugrid[ptr2d] = sqrtf(1 - d2);

}

// calculate the flux at each grid-time point pair
__global__ void populate_flux_points(
    const double * xc,
    const double * yc,
    const int n_pos_elem,
    const double transp,
    const double r0,
    const double rmaj,
    const double rmin,
    const double pa,
    const double alimb,
    const double blimb,
    const double * xgrid,
    const double * ygrid,
    const double * mugrid,
    const int n_grid_elem,
    double * lc
){
    const int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (x_idx >= n_pos_elem || y_idx >= n_grid_elem) return;
}